
#include <hip/hip_runtime.h>
#include <stdio.h>
#define COUNT 40

enum FizzBuzzEnum {
    FIZZBUZZ, FIZZ, BUZZ, NONE
};

__device__ FizzBuzzEnum to_enum(int num) {
    return (num%15)==0 ? FIZZBUZZ : (num%3)==0 ? FIZZ : (num%5)==0 ? BUZZ : NONE;
}
__global__ void dev_fizzbuzz(FizzBuzzEnum *d_fizzbuzz) {
    int num = threadIdx.x + 1;
    d_fizzbuzz[num] = to_enum(num);
}

int main() {
    FizzBuzzEnum h_fizzbuzz[COUNT];
    FizzBuzzEnum *d_fizzbuzz;

    hipMalloc(&d_fizzbuzz, sizeof(h_fizzbuzz));

    dev_fizzbuzz<<<1,COUNT>>>(d_fizzbuzz);

    hipMemcpy(h_fizzbuzz, d_fizzbuzz, sizeof(h_fizzbuzz), hipMemcpyDeviceToHost);

    for (int i=0;i<COUNT;++i) {
	switch(h_fizzbuzz[i]) {
	    case FIZZBUZZ:
		puts("FIZZBUZZ");
		break;
	    case FIZZ:
		puts("FIZZ");
		break;
	    case BUZZ:
		puts("BUZZ");
		break;
	    case NONE:
		printf("%d\n", i+1);
		break;
	}
    }

    hipFree(d_fizzbuzz);

    return 0;
}
// vim: set ft=cpp:
